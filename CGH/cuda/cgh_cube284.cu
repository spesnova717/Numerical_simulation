#include <stdio.h>
#include <math.h>
#include<stdint.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>


#define WID 1024
#define HEI 1024

#pragma pack(push,1)
typedef struct tagBITMAPFILEHEADER
{
  unsigned short bfType;
  uint32_t  bfSize;
  unsigned short bfReserved1;
  unsigned short bfReserved2;
  uint32_t  bf0ffBits;
}BITMAPFILEHEADER;
#pragma pack(pop)

typedef struct tagBITMAPINFOHEADER
{
  uint32_t  biSize;
  int32_t biWidth;
  int32_t biHeight;
  unsigned short  biPlanes;
  unsigned short  biBitCount;
  uint32_t   biCompression;
  uint32_t   biSizeImage;
  int32_t  biXPelsPerMeter;
  int32_t  biYPelsPerMeter;
  uint32_t   biCirUsed;
  uint32_t   biCirImportant;
}BITMAPINFOHEADER;

typedef struct tagRGBQUAD
{
  unsigned char  rgbBlue;
  unsigned char  rgbGreen;
  unsigned char  rgbRed;
  unsigned char  rgbReserved;
}RGBQUAD;

typedef struct tagBITMAPINFO
{
  BITMAPINFOHEADER bmiHeader;
  RGBQUAD          bmiColors[1];
}BITMAPINFO;


__global__ void distance_gpu(int *x_d,int *y_d,double *z_d,double *img_buf_d,int *tensuu_d)
{
  int i,j,k;

  i=blockIdx.x*128+threadIdx.x;

  double kankaku,hatyou,goukei;

  hatyou=0.633;
  kankaku=10.5;
  goukei=2.0*M_PI*kankaku/hatyou;

  for(j=0;j<WID;j++){
    for(k=0;k<*tensuu_d;k++){
      img_buf_d[i*WID+j]=img_buf_d[i*WID+j]+cos(goukei*sqrt((j-x_d[k])*(j-x_d[k])+(i-y_d[k])*(i-y_d[k])+z_d[k]*z_d[k]));
    }
  }
}


int main(){

    int tensuu;

    BITMAPFILEHEADER    BmpFileHeader;
    BITMAPINFOHEADER    BmpInfoHeader;
    RGBQUAD             RGBQuad[256];

    FILE *fp;
    int i,j;

    BmpFileHeader.bfType                =19778;
    BmpFileHeader.bfSize                =14+40+1024+(WID*HEI);
    BmpFileHeader.bfReserved1           =0;
    BmpFileHeader.bfReserved2           =0;
    BmpFileHeader.bf0ffBits             =14+40+1024;

    BmpInfoHeader.biSize                =40;
    BmpInfoHeader.biWidth               =WID;
    BmpInfoHeader.biHeight              =HEI;
    BmpInfoHeader.biPlanes              =1;
    BmpInfoHeader.biBitCount            =8;     //256階調
    BmpInfoHeader.biCompression         =0L;
    BmpInfoHeader.biSizeImage           =0L;
    BmpInfoHeader.biXPelsPerMeter       =0L;
    BmpInfoHeader.biYPelsPerMeter       =0L;
    BmpInfoHeader.biCirUsed             =0L;
    BmpInfoHeader.biCirImportant        =0L;

    for(i=0;i<256;i++){
       RGBQuad[i].rgbBlue                =i;
       RGBQuad[i].rgbGreen               =i;
       RGBQuad[i].rgbRed                 =i;
       RGBQuad[i].rgbReserved            =0;
    }

    char filename[20]={};
    printf("Filename:  ");
    //scanf("%s",filename);

    //fp=fopen(filename,"rb");
    fp=fopen("cube284.3d","rb");
    if(fp==NULL){
      printf("File Open ERROR\n");
    }

    fread(&tensuu,sizeof(int),1,fp);
    printf("num=%d\n",tensuu);

    int x[tensuu];
    int y[tensuu];
    double z[tensuu];

    int *tensuu_d;

    hipMalloc((void**)&tensuu_d,sizeof(int));
    hipMemcpy(tensuu_d,&tensuu,sizeof(int),hipMemcpyHostToDevice);

    int *x_d,*y_d;
    double *z_d;
    double *img_buf_d;

    dim3 blocks(8,1,1);
    dim3 threads(128,1,1);

    int x_buf,y_buf,z_buf;

    for(i=0;i<tensuu;i++){
      fread(&x_buf,sizeof(int),1,fp);
      fread(&y_buf,sizeof(int),1,fp);
      fread(&z_buf,sizeof(int),1,fp);

      x[i]=x_buf*40+512;
      y[i]=y_buf*40+512;
      z[i]=((double)z_buf)*40+100000.0;
    }
    fclose(fp);


    hipMalloc((void**)&x_d,tensuu*sizeof(int));
    hipMalloc((void**)&y_d,tensuu*sizeof(int));
    hipMalloc((void**)&z_d,tensuu*sizeof(double));

    hipMalloc((void**)&img_buf_d,WID*HEI*sizeof(double));

    double *img_buf;

    img_buf=(double *)malloc(sizeof(double)*WID*HEI);
    for(i=0;i<WID*HEI;i++){
      img_buf[i]=0.0;
    }

    hipMemcpy(x_d,x,tensuu*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(y_d,y,tensuu*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(z_d,z,tensuu*sizeof(double),hipMemcpyHostToDevice);

    hipMemcpy(img_buf_d,img_buf,WID*HEI*sizeof(double),hipMemcpyHostToDevice);

    distance_gpu<<<blocks,threads>>>(x_d,y_d,z_d,img_buf_d,tensuu_d);

    hipMemcpy(img_buf,img_buf_d,WID*HEI*sizeof(double),hipMemcpyDeviceToHost);




    double min,max,mid;

    min=img_buf[0];
    max=img_buf[0];

    for(i=0;i<HEI;i++){
      for(j=0;j<WID;j++){
        if(min>img_buf[i*WID+j]){
          min=img_buf[i*WID+j];
        }
        if(max<img_buf[i*WID+j]){
          max=img_buf[i*WID+j];
        }
      }
    }

    mid=0.5*(min+max);

    printf("min = %lf  max = %lf  mid = %lf\n",min,max,mid);


    unsigned char *img;
    img=(unsigned char *)malloc(sizeof(unsigned char)*WID*HEI);

    for(i=0;i<WID*HEI;i++){
      if(img_buf[i]<mid){
        img[i]=0;
      }
      if(img_buf[i]>mid){
        img[i]=255;
      }
    }

    FILE *fp1;
    fp1=fopen("cgh_root_gpu.bmp","wb");
    if(fp1==NULL){
      printf("ファイルオープンエラー\n");
    }

    fwrite(&BmpFileHeader, sizeof(BmpFileHeader) , 1 ,fp1);
    fwrite(&BmpInfoHeader, sizeof(BmpInfoHeader) , 1 ,fp1);
    fwrite(&RGBQuad[0], sizeof(RGBQuad[0]) , 256 ,fp1);
    fwrite(img,sizeof(unsigned char),WID*HEI,fp1);

    free(img);
    free(img_buf);
    fclose(fp1);

    hipFree(tensuu_d);
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
    hipFree(img_buf_d);
    

    return 0;



}
