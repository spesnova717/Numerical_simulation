#include <stdio.h>
#include <math.h>
#include <stdint.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define WID 1024
#define HEI 1024

#pragma pack(push,1)
typedef struct tagBITMAPFILEHEADER
{
  unsigned short bfType;
  uint32_t  bfSize;
  unsigned short bfReserved1;
  unsigned short bfReserved2;
  uint32_t  bf0ffBits;
}BITMAPFILEHEADER;
#pragma pack(pop)

typedef struct tagBITMAPINFOHEADER
{
  uint32_t  biSize;
  int32_t biWidth;
  int32_t biHeight;
  unsigned short  biPlanes;
  unsigned short  biBitCount;
  uint32_t   biCompression;
  uint32_t   biSizeImage;
  int32_t  biXPelsPerMeter;
  int32_t  biYPelsPerMeter;
  uint32_t   biCirUsed;
  uint32_t   biCirImportant;
}BITMAPINFOHEADER;

typedef struct tagRGBQUAD
{
  unsigned char  rgbBlue;
  unsigned char  rgbGreen;
  unsigned char  rgbRed;
  unsigned char  rgbReserved;
}RGBQUAD;

typedef struct tagBITMAPINFO
{
  BITMAPINFOHEADER bmiHeader;
  RGBQUAD          bmiColors[1];
}BITMAPINFO;


__global__ void distance_gpu(int *x_d,int *y_d,float *z_d,float *img_buf_d,int *tensuu_d)
{
  int i,j,k;

  i=blockIdx.x*128+threadIdx.x;

  float kankaku,hatyou,goukei,pi;

  hatyou=0.633F;
  kankaku=10.5F;
  pi=3.14159265F;
  goukei=2.0F*pi*kankaku/hatyou;

  float dx,dy,tmp;

  for(j=0;j<WID;j++){
    tmp=0.0F;
    for(k=0;k<*tensuu_d;k++){
      dx=(float)(x_d[k]-j);
      dy=(float)(y_d[k]-i);
      tmp=tmp+cos(goukei*0.5F*(dx*dx+dy*dy)/z_d[k]);
    }
  img_buf_d[i*WID+j] = tmp;
  }
}


int main(){

    int tensuu;

    BITMAPFILEHEADER    BmpFileHeader;
    BITMAPINFOHEADER    BmpInfoHeader;
    RGBQUAD             RGBQuad[256];

    FILE *fp;
    int i,j;

    BmpFileHeader.bfType                =19778;
    BmpFileHeader.bfSize                =14+40+1024+(WID*HEI);
    BmpFileHeader.bfReserved1           =0;
    BmpFileHeader.bfReserved2           =0;
    BmpFileHeader.bf0ffBits             =14+40+1024;

    BmpInfoHeader.biSize                =40;
    BmpInfoHeader.biWidth               =WID;
    BmpInfoHeader.biHeight              =HEI;
    BmpInfoHeader.biPlanes              =1;
    BmpInfoHeader.biBitCount            =8;     //256階調
    BmpInfoHeader.biCompression         =0L;
    BmpInfoHeader.biSizeImage           =0L;
    BmpInfoHeader.biXPelsPerMeter       =0L;
    BmpInfoHeader.biYPelsPerMeter       =0L;
    BmpInfoHeader.biCirUsed             =0L;
    BmpInfoHeader.biCirImportant        =0L;

    for(i=0;i<256;i++){
       RGBQuad[i].rgbBlue                =i;
       RGBQuad[i].rgbGreen               =i;
       RGBQuad[i].rgbRed                 =i;
       RGBQuad[i].rgbReserved            =0;
    }

    char filename[20]={};
    //printf("ファイル名を入力してください :  ");
    //scanf("%s",filename);

    //fp=fopen(filename,"rb");
    fp=fopen("cubex.3d","rb");
    if(fp==NULL){
      printf("File Open ERROR\n");
    }

    fread(&tensuu,sizeof(int),1,fp);
    printf("num=%d\n",tensuu);

    int x[tensuu];
    int y[tensuu];
    float z[tensuu];

    int *tensuu_d;

    hipMalloc((void**)&tensuu_d,sizeof(int));
    hipMemcpy(tensuu_d,&tensuu,sizeof(int),hipMemcpyHostToDevice);

    int *x_d,*y_d;
    float *z_d;
    float *img_buf_d;

    dim3 blocks(8,1,1);
    dim3 threads(128,1,1);

    int x_buf,y_buf,z_buf;

    for(i=0;i<tensuu;i++){
      fread(&x_buf,sizeof(int),1,fp);
      fread(&y_buf,sizeof(int),1,fp);
      fread(&z_buf,sizeof(int),1,fp);

      x[i]=x_buf*40+512;
      y[i]=y_buf*40+512;
      z[i]=((float)z_buf)*40+100000.0F;
    }
    fclose(fp);


    hipMalloc((void**)&x_d,tensuu*sizeof(int));
    hipMalloc((void**)&y_d,tensuu*sizeof(int));
    hipMalloc((void**)&z_d,tensuu*sizeof(float));

    hipMalloc((void**)&img_buf_d,WID*HEI*sizeof(float));

    float *img_buf;

    img_buf=(float *)malloc(sizeof(float)*WID*HEI);
    for(i=0;i<WID*HEI;i++){
      img_buf[i]=0.0F;
    }

    hipMemcpy(x_d,x,tensuu*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(y_d,y,tensuu*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(z_d,z,tensuu*sizeof(float),hipMemcpyHostToDevice);

    hipMemcpy(img_buf_d,img_buf,WID*HEI*sizeof(float),hipMemcpyHostToDevice);

    distance_gpu<<<blocks,threads>>>(x_d,y_d,z_d,img_buf_d,tensuu_d);

    hipMemcpy(img_buf,img_buf_d,WID*HEI*sizeof(float),hipMemcpyDeviceToHost);




    float min,max,mid;

    min=img_buf[0];
    max=img_buf[0];

    for(i=0;i<HEI;i++){
      for(j=0;j<WID;j++){
        if(min>img_buf[i*WID+j]){
          min=img_buf[i*WID+j];
        }
        if(max<img_buf[i*WID+j]){
          max=img_buf[i*WID+j];
        }
      }
    }

    mid=0.5F*(min+max);

    printf("min = %lf  max = %lf  mid = %lf\n",min,max,mid);


    unsigned char *img;
    img=(unsigned char *)malloc(sizeof(unsigned char)*WID*HEI);

    for(i=0;i<WID*HEI;i++){
      if(img_buf[i]<mid){
        img[i]=0;
      }
      if(img_buf[i]>mid){
        img[i]=255;
      }
    }

    FILE *fp1;
    fp1=fopen("cgh_root_gpu.bmp","wb");
    if(fp1==NULL){
      printf("ファイルオープンエラー\n");
    }

    fwrite(&BmpFileHeader, sizeof(BmpFileHeader) , 1 ,fp1);
    fwrite(&BmpInfoHeader, sizeof(BmpInfoHeader) , 1 ,fp1);
    fwrite(&RGBQuad[0], sizeof(RGBQuad[0]) , 256 ,fp1);
    fwrite(img,sizeof(unsigned char),WID*HEI,fp1);

    free(img);
    free(img_buf);
    fclose(fp1);

    hipFree(tensuu_d);
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
    hipFree(img_buf_d);


    return 0;

}
